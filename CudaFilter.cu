#include "hip/hip_runtime.h"
/*
 * $Id: CudaFilter.cu,v 1.7 2011-04-26 06:39:19 ueshiba Exp $
 */
#include "TU/CudaFilter.h"
#include "TU/CudaUtility.h"
#include <boost/mpl/size_t.hpp>

namespace TU
{
/************************************************************************
*  global constatnt variables						*
************************************************************************/
static const size_t		BlockDimX = 32;
static const size_t		BlockDimY = 16;
    
static __constant__ float	_lobeH[CudaFilter2::LOBE_SIZE_MAX];
static __constant__ float	_lobeV[CudaFilter2::LOBE_SIZE_MAX];

/************************************************************************
*  device functions							*
************************************************************************/
static inline __device__ float
convolve(const float* in_s, const float* lobe, boost::mpl::size_t<17>)
{
  // ����Ĺ��17���Ǥζ��ؿ����߹��ߥ����ͥ�
    return lobe[ 0] * (in_s[-16] + in_s[16])
	 + lobe[ 1] * (in_s[-15] + in_s[15])
	 + lobe[ 2] * (in_s[-14] + in_s[14])
	 + lobe[ 3] * (in_s[-13] + in_s[13])
	 + lobe[ 4] * (in_s[-12] + in_s[12])
	 + lobe[ 5] * (in_s[-11] + in_s[11])
	 + lobe[ 6] * (in_s[-10] + in_s[10])
	 + lobe[ 7] * (in_s[ -9] + in_s[ 9])
	 + lobe[ 8] * (in_s[ -8] + in_s[ 8])
	 + lobe[ 9] * (in_s[ -7] + in_s[ 7])
	 + lobe[10] * (in_s[ -6] + in_s[ 6])
	 + lobe[11] * (in_s[ -5] + in_s[ 5])
	 + lobe[12] * (in_s[ -4] + in_s[ 4])
	 + lobe[13] * (in_s[ -3] + in_s[ 3])
	 + lobe[14] * (in_s[ -2] + in_s[ 2])
	 + lobe[15] * (in_s[ -1] + in_s[ 1])
	 + lobe[16] *  in_s[  0];
}
    
static inline __device__ float
convolve(const float* in_s, const float* lobe, boost::mpl::size_t<16>)
{
  // ����Ĺ��16���Ǥδ�ؿ����߹��ߥ����ͥ�
    return lobe[ 0] * (in_s[-16] - in_s[16])
	 + lobe[ 1] * (in_s[-15] - in_s[15])
	 + lobe[ 2] * (in_s[-14] - in_s[14])
	 + lobe[ 3] * (in_s[-13] - in_s[13])
	 + lobe[ 4] * (in_s[-12] - in_s[12])
	 + lobe[ 5] * (in_s[-11] - in_s[11])
	 + lobe[ 6] * (in_s[-10] - in_s[10])
	 + lobe[ 7] * (in_s[ -9] - in_s[ 9])
	 + lobe[ 8] * (in_s[ -8] - in_s[ 8])
	 + lobe[ 9] * (in_s[ -7] - in_s[ 7])
	 + lobe[10] * (in_s[ -6] - in_s[ 6])
	 + lobe[11] * (in_s[ -5] - in_s[ 5])
	 + lobe[12] * (in_s[ -4] - in_s[ 4])
	 + lobe[13] * (in_s[ -3] - in_s[ 3])
	 + lobe[14] * (in_s[ -2] - in_s[ 2])
	 + lobe[15] * (in_s[ -1] - in_s[ 1]);
}
    
static inline __device__ float
convolve(const float* in_s, const float* lobe, boost::mpl::size_t<9>)
{
  // ����Ĺ��9���Ǥζ��ؿ����߹��ߥ����ͥ�
    return lobe[0] * (in_s[-8] + in_s[8])
	 + lobe[1] * (in_s[-7] + in_s[7])
	 + lobe[2] * (in_s[-6] + in_s[6])
	 + lobe[3] * (in_s[-5] + in_s[5])
	 + lobe[4] * (in_s[-4] + in_s[4])
	 + lobe[5] * (in_s[-3] + in_s[3])
	 + lobe[6] * (in_s[-2] + in_s[2])
	 + lobe[7] * (in_s[-1] + in_s[1])
	 + lobe[8] *  in_s[ 0];
}
    
static inline __device__ float
convolve(const float* in_s, const float* lobe, boost::mpl::size_t<8>)
{
  // ����Ĺ��8���Ǥδ�ؿ����߹��ߥ����ͥ�
    return lobe[0] * (in_s[-8] - in_s[8])
	 + lobe[1] * (in_s[-7] - in_s[7])
	 + lobe[2] * (in_s[-6] - in_s[6])
	 + lobe[3] * (in_s[-5] - in_s[5])
	 + lobe[4] * (in_s[-4] - in_s[4])
	 + lobe[5] * (in_s[-3] - in_s[3])
	 + lobe[6] * (in_s[-2] - in_s[2])
	 + lobe[7] * (in_s[-1] - in_s[1]);
}
    
static inline __device__ float
convolve(const float* in_s, const float* lobe, boost::mpl::size_t<5>)
{
  // ����Ĺ��5���Ǥζ��ؿ����߹��ߥ����ͥ�
    return lobe[0] * (in_s[-4] + in_s[4])
	 + lobe[1] * (in_s[-3] + in_s[3])
	 + lobe[2] * (in_s[-2] + in_s[2])
	 + lobe[3] * (in_s[-1] + in_s[1])
	 + lobe[4] *  in_s[ 0];
}
    
static inline __device__ float
convolve(const float* in_s, const float* lobe, boost::mpl::size_t<4>)
{
  // ����Ĺ��4���Ǥδ�ؿ����߹��ߥ����ͥ�
    return lobe[0] * (in_s[-4] - in_s[4])
	 + lobe[1] * (in_s[-3] - in_s[3])
	 + lobe[2] * (in_s[-2] - in_s[2])
	 + lobe[3] * (in_s[-1] - in_s[1]);
}
    
static inline __device__ float
convolve(const float* in_s, const float* lobe, boost::mpl::size_t<3>)
{
  // ����Ĺ��3���Ǥζ��ؿ����߹��ߥ����ͥ�
    return lobe[0] * (in_s[-2] + in_s[2])
	 + lobe[1] * (in_s[-1] + in_s[1])
	 + lobe[2] *  in_s[ 0];
}
    
static inline __device__ float
convolve(const float* in_s, const float* lobe, boost::mpl::size_t<2>)
{
  // ����Ĺ��2���Ǥδ�ؿ����߹��ߥ����ͥ�
    return lobe[0] * (in_s[-2] - in_s[2])
	 + lobe[1] * (in_s[-1] - in_s[1]);    
}
    
template <size_t L, class S, class T> static __global__ void
filterH_kernel(const S* in, T* out, uint stride_i, uint stride_o)
{
    const int	x   = blockIdx.x*blockDim.x + threadIdx.x,
		y   = blockIdx.y*blockDim.y + threadIdx.y,
		xy  = y*stride_i + x,
    		dxy = blockDim.x;

  // in_s[]���:blockDim.y, ��:3*blockDim.x ��2��������Ȥ��ư�����
    const int	xy_s  = threadIdx.y*(3*blockDim.x) + blockDim.x + threadIdx.x,
		dxy_s = blockDim.x;

  // ��������3�ĤΥ�����(����åɥ֥�å����б�)��ͭ����˥��ԡ�
    __shared__ float	in_s[BlockDimX * (3*BlockDimY + 1)];
    in_s[xy_s - dxy_s] = in[xy - dxy];
    in_s[xy_s	     ] = in[xy	    ];
    in_s[xy_s + dxy_s] = in[xy + dxy];
    __syncthreads();
    
  // ���±黻
    out[y*stride_o + x] = convolve(in_s + xy_s, _lobeH,
				   boost::mpl::size_t<L>());
}
    
template <size_t L, class S, class T> static __global__ void
filterV_kernel(const S* in, T* out, uint stride_i, uint stride_o)
{
    const int	x   = blockIdx.x*blockDim.x + threadIdx.x,
		y   = blockIdx.y*blockDim.y + threadIdx.y,
		xy  = y*stride_i + x,
		dxy = blockDim.y*stride_i;

  // bank conflict���ɤ����ᡤin_s[]���:blockDim.x, ��:3*blockDim.y + 1 ��
  // 2��������Ȥ��ư�����
    const int	xy_s  = threadIdx.x*(3*blockDim.y + 1)
		      + blockDim.y + threadIdx.y,
		dxy_s = blockDim.y;
    
  // ��������3�ĤΥ�����(����åɥ֥�å����б�)��ͭ����˥��ԡ�
    __shared__ float	in_s[BlockDimX * (3*BlockDimY + 1)];
    in_s[xy_s - dxy_s] = in[xy - dxy];
    in_s[xy_s	     ] = in[xy	    ];
    in_s[xy_s + dxy_s] = in[xy + dxy];
    __syncthreads();
    
  // ���±黻
    out[y*stride_o + x] = convolve(in_s + xy_s, _lobeH,
				   boost::mpl::size_t<L>());
}

/************************************************************************
*  static functions							*
************************************************************************/
template <size_t L, class S, class T> inline static void
convolveH_dispatch(const CudaArray2<S>& in, CudaArray2<T>& out)
{
    const size_t	lobeSize = L & ~0x1;	// �濴����ޤޤʤ�����Ĺ

  // ����
    int		xs = lobeSize;
    dim3	threads(lobeSize, BlockDimY);
    dim3	blocks((BlockDimX - xs) / threads.x, 1);
    filterH_kernel<L><<<blocks, threads>>>(in[0].data()  + xs,
					   out[0].data() + xs,
					   in.stride(), out.stride());
    xs += blocks.x * threads.x;

  // ����
    threads.x = BlockDimX;
    blocks.x  = (out.stride() - xs) / threads.x;
    filterH_kernel<L><<<blocks, threads>>>(in[0].data()  + xs,
					   out[0].data() + xs,
					   in.stride(), out.stride());
    int		ys = blocks.y * threads.y;
    if (ys >= in.nrow())
	return;

  // ���
    blocks.x = out.stride() / threads.x;
    blocks.y = (out.nrow() - ys) / threads.y;
    filterH_kernel<L><<<blocks, threads>>>(in[ys].data()  + xs,
					   out[ys].data() + xs,
					   in.stride(), out.stride());
    ys += blocks.y * threads.y;
    if (ys >= in.nrow())
	return;

  // ����
    blocks.x  = (out.stride() - lobeSize) / threads.x;
    threads.y = out.nrow() - ys;
    blocks.y  = 1;
    filterH_kernel<L><<<blocks, threads>>>(in[ys].data(), out[ys].data(),
					   in.stride(), out.stride());
    xs = blocks.x * threads.x;

  // ����
    threads.x = lobeSize;
    blocks.x  = (out.stride() - lobeSize - xs) / threads.x;
    filterH_kernel<L><<<blocks, threads>>>(in[ys].data()  + xs,
					   out[ys].data() + xs,
					   in.stride(), out.stride());
}
    
template <size_t L, class S, class T> inline static void
convolveV_dispatch(const CudaArray2<S>& in, CudaArray2<T>& out)
{
    const size_t	lobeSize = L & ~0x1;	// �濴����ޤޤʤ�����Ĺ

  // �ǽ��BlockDimY�ԡʺǽ��lobeSize�Ԥ������
    int		ys = lobeSize;
    dim3	threads(BlockDimX, lobeSize);
    dim3	blocks(out.stride() / threads.x, (BlockDimY - ys) / threads.y);
    filterV_kernel<L><<<blocks, threads>>>(in[ys].data(), out[ys].data(),
					   in.stride(), out.stride());
    ys += blocks.y * threads.y;
    if (ys >= in.nrow())
	return;
    
  // BlockDimY�԰ʾ夬�Ĥ�褦�˽���������åɿ���BlockDimY�ˤ��ƽ���
    threads.y = BlockDimY;
    blocks.y  = (out.nrow() - ys) / threads.y - 1;
    filterV_kernel<L><<<blocks, threads>>>(in[ys].data(), out[ys].data(),
					   in.stride(), out.stride());
    ys += blocks.y * threads.y;
    if (ys >= in.nrow())
	return;
    
  // �Ĥ�Ͻ���������åɿ���lobeSize�ˤ��ƽ����ʺǸ��lobeSize�Ԥ������
    threads.y = lobeSize;
    blocks.y  = (out.nrow() - ys - 1) / threads.y;
    ys = out.nrow() - (1 + blocks.y) * threads.y;
    filterV_kernel<L><<<blocks, threads>>>(in[ys].data(), out[ys].data(),
					   in.stride(), out.stride());
}
    
/************************************************************************
*  class CudaFilter2							*
************************************************************************/
//! CUDA�ˤ��2�����ե��륿���������롥
CudaFilter2::CudaFilter2()
    :_lobeSizeH(0), _lobeSizeV(0)
{
    int	device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&_prop, device);
}

//! 2�����ե��륿�Υ��֤����ꤹ�롥
/*!
  Ϳ������֤�Ĺ���ϡ����߹��ߥ����ͥ뤬���ؿ��ξ��2^n + 1, ��ؿ��ξ��2^n
  (n = 1, 2, 3, 4)�Ǥʤ���Фʤ�ʤ���
  \param lobeH	����������
  \param lobeV	����������
  \return	����2�����ե��륿
*/
CudaFilter2&
CudaFilter2::initialize(const Array<float>& lobeH, const Array<float>& lobeV)
{
    using namespace	std;
    
    if (_lobeSizeH > LOBE_SIZE_MAX || _lobeSizeV > LOBE_SIZE_MAX)
	throw runtime_error("CudaFilter2::initialize: too large lobe size!");
    
    _lobeSizeH = lobeH.size();
    _lobeSizeV = lobeV.size();
    hipMemcpyToSymbol(HIP_SYMBOL(_lobeH), lobeH.data(), lobeH.size()*sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(_lobeV), lobeV.data(), lobeV.size()*sizeof(float));

    return *this;
}
    
//! Ϳ����줿2��������Ȥ��Υե��륿����߹���
/*!
  \param in	����2��������
  \param out	����2��������
  \return	���Υե��륿����
*/
template <class S, class T> const CudaFilter2&
CudaFilter2::convolve(const CudaArray2<S>& in, CudaArray2<T>& out) const
{
    using namespace	std;

  // �������˾��߹��ࡥ
    _buf.resize(in.nrow(), in.ncol());

    switch (_lobeSizeH)
    {
      case 17:
	convolveH_dispatch<17>(in, _buf);
	break;
      case 16:
	convolveH_dispatch<16>(in, _buf);
	break;
      case  9:
	convolveH_dispatch< 9>(in, _buf);
	break;
      case  8:
	convolveH_dispatch< 9>(in, _buf);
	break;
      case  5:
	convolveH_dispatch< 5>(in, _buf);
	break;
      case  4:
	convolveH_dispatch< 4>(in, _buf);
	break;
      case  3:
	convolveH_dispatch< 3>(in, _buf);
	break;
      case  2:
	convolveH_dispatch< 2>(in, _buf);
	break;
      default:
	throw runtime_error("CudaFilter2::convolve: unsupported horizontal lobe size!");
    }

  // �������˾��߹��ࡥ
    out.resize(_buf.nrow(), _buf.ncol());
    
    switch (_lobeSizeV)
    {
      case 17:
	convolveV_dispatch<17>(_buf, out);
	break;
      case 16:
	convolveV_dispatch<16>(_buf, out);
	break;
      case  9:
	convolveV_dispatch< 9>(_buf, out);
	break;
      case  8:
	convolveV_dispatch< 8>(_buf, out);
	break;
      case  5:
	convolveV_dispatch< 5>(_buf, out);
	break;
      case  4:
	convolveV_dispatch< 4>(_buf, out);
	break;
      case  3:
	convolveV_dispatch< 3>(_buf, out);
	break;
      case  2:
	convolveV_dispatch< 2>(_buf, out);
	break;
      default:
	throw runtime_error("CudaFilter2::convolve: unsupported vertical lobe size!");
    }

    return *this;
}

template const CudaFilter2&
CudaFilter2::convolve(const CudaArray2<u_char>& in,
			    CudaArray2<u_char>& out)		const	;
template const CudaFilter2&
CudaFilter2::convolve(const CudaArray2<u_char>& in,
			    CudaArray2<float>&  out)		const	;
template const CudaFilter2&
CudaFilter2::convolve(const CudaArray2<float>& in,
			    CudaArray2<u_char>& out)		const	;
template const CudaFilter2&
CudaFilter2::convolve(const CudaArray2<float>& in,
			    CudaArray2<float>& out)		const	;
}
