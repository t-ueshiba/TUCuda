#include "hip/hip_runtime.h"
#include "TU/cu/Array++.h"
#include "TU/cu/array.h"

namespace TU
{
namespace cu
{
namespace device
{
__global__ void
printTest()
{
    printf("tid=%d\n", threadIdx.x);
}

template <class T, size_t D> __global__ void
arrayTest(array<T, D> a)
{
    const int	i = threadIdx.x;

    printf("tid=%d\n", i);

    if (i < a.size())
	printf("%f\n", a[i]);
}

}	// namespace device

template <class T, size_t D> void
doJob()
{
    std::cerr << "OK" << std::endl;

    Array<device::array<T, D> >	A(1);
    auto&		a = A[0];
    for (size_t i = 0; i < a.size(); ++i)
	a[i] = i;
    device::arrayTest<<<1, D>>>(a);
}

}	// namespace cu
}	// namespace TU

int
main()
{
    TU::cu::doJob<float, 3>();

    return 0;
}
