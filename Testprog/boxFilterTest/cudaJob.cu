#include "hip/hip_runtime.h"
/*
 *  $Id$
 */
#include "TU/Image++.h"
#include "TU/Profiler.h"
#if 1
#  include "TU/cuda/BoxFilter.h"
#elif 0
#  include "TU/cuda/NewBoxFilter.h"
#else
#  include "TU/cuda/NeoBoxFilter.h"
#endif
#include "TU/cuda/chrono.h"

namespace TU
{
template <class S, class T> void
cudaJob(const Image<S>& in, Image<T>& out, size_t winSize)
{
    cuda::BoxFilter2<T, 15>	cudaFilter(winSize, winSize);
    cuda::Array2<S>		in_d(in.nrow(), in.ncol(), 32);
    in_d = in;
    cuda::Array2<T>		out_d(in_d.nrow(), in_d.ncol(), 32);
    cudaFilter.convolve(in_d.cbegin(), in_d.cend(), out_d.begin());
    hipDeviceSynchronize();

    Profiler<cuda::clock>	cudaProfiler(1);
    constexpr size_t		NITER = 1000;
    for (size_t n = 0; n < NITER; ++n)
    {
	cudaProfiler.start(0);
	cudaFilter.convolve(in_d.cbegin(), in_d.cend(), out_d.begin());
	cudaProfiler.nextFrame();
    }
    cudaProfiler.print(std::cerr);

    out_d.write(out);
}

template void
cudaJob(const Image<float>& in, Image<float>& out, size_t winSize)	;
template void
cudaJob(const Image<u_char>& in, Image<short>& out, size_t winSize)	;
    
}
