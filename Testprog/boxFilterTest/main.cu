#include "hip/hip_runtime.h"
/*
 *  $Id: main.cc,v 1.1 2012-08-30 00:13:51 ueshiba Exp $
 */
#include "TU/Image++.h"

namespace TU
{
template <class U, class S, class T> void
cudaJob(const Array2<S>& in, Array2<T>& out, size_t winSize)	;

template <class S, class T> void
cpuJob(const Array2<S>& in, Array2<T>& out, size_t winSize)	;
}

/************************************************************************
*  Global fucntions							*
************************************************************************/
int
main(int argc, char *argv[])
{
    using namespace	std;
    using namespace	TU;
#if 0
    using in_t	= u_char;
    using mid_t	= float;
    using out_t	= float;
#else
    using in_t	= RGBA;
    using mid_t = float4;
    using out_t	= RGBA;
#endif    
    size_t		winSize = 3;
    extern char*	optarg;
    for (int c; (c = getopt(argc, argv, "w:")) != -1; )
	switch (c)
	{
	  case 'w':
	    winSize = atoi(optarg);
	    break;
	}
    
    try
    {
	Image<in_t>	in;
	in.restore(cin);				// 原画像を読み込む

      // GPUによって計算する．
	Image<out_t>	out(in.width(), in.height());
	cudaJob<mid_t>(in, out, winSize);
	out.save(cout);					// 結果画像をセーブ

      // CPUによって計算する．
	cpuJob(in, out, winSize);
	out.save(cout);
    }
    catch (exception& err)
    {
	cerr << err.what() << endl;
	return 1;
    }

    return 0;
}
