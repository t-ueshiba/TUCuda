#include "hip/hip_runtime.h"
/*
 * $Id: cuda_image_interpolate.cu,v 1.3 2009-04-20 01:16:37 ueshiba Exp $
 */
#include "TU/CudaDeviceMemory.h"
#include "TU/Image++.h"

#include "cuda_image_interpolate_kernel.h"

namespace TU
{
/*
 *  �����ϰ����η��� (const) Image<RGBA>& �Ȥ������Ȥ���Ǥ��뤬��
 *  CUDA-2.1�� nvcc �ǥ���ѥ��뤷��C++�ؿ��ϲ��ۥ��дؿ������
 *  ���饹�Υ��֥������Ȥ������������ʤ��褦�Ǥ��롥
 */ 
void
interpolate(const Array2<ImageLine<RGBA> >& image0,
	    const Array2<ImageLine<RGBA> >& image1,
		  Array2<ImageLine<RGBA> >& image2)
{
    using namespace	std;

  // timer
    u_int	timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutStartTimer(timer));

  // allocate device memory and copy host memory to them
    CudaDeviceMemory2<uchar4>	d_image0, d_image1, d_image2;
    d_image0.readFrom(image0);
    d_image1.readFrom(image1);
    d_image2.resize(image0.nrow(), image0.ncol());
    
  // setup execution parameters
    dim3  threads(16, 16, 1);
    dim3  blocks(image0.ncol()/threads.x, image0.nrow()/threads.y, 1);
    cerr << blocks.x << 'x' << blocks.y << " blocks..." << endl;
    
  // execute the kernel
    cerr << "Let's go!" << endl;
    for (int i = 0; i < 1000; ++i)
	interpolate_kernel<<<blocks, threads>>>((const uchar4*)d_image0,
						(const uchar4*)d_image1,
						(      uchar4*)d_image2,
						d_image2.ncol(),
						d_image2.nrow(), 0.5f);
    cerr << "Returned!" << endl;
    
  // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

  // copy result from device to host
    d_image2.writeTo(image2);

  // time
    CUT_SAFE_CALL(cutStopTimer(timer));
    cerr << "Processing time: " << cutGetTimerValue(timer) << " (ms)" << endl;
    CUT_SAFE_CALL(cutDeleteTimer(timer));

#if 0	
  // compute reference solution
    float* reference = (float*) malloc(mem_size);
    computeGold(reference, h_idata, num_threads);

  // check result
    if (cutCheckCmdLineFlag(argc, ( const char** ) argv, "regression" ))
    {
      // write file for regression test
        CUT_SAFE_CALL(cutWriteFilef( "./data/regression.dat",
				     h_odata, num_threads, 0.0));
    }
    else
    {
        CUTBoolean res = cutComparef(reference, h_odata, num_threads);
        printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");
    }
#endif
}

}
