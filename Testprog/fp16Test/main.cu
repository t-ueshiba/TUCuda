#include "hip/hip_runtime.h"
/*
 * $Id$
 */
#include "TU/cu/Array++.h"
#include "TU/cu/fp16.h"

namespace TU
{
namespace cu
{
namespace device
{
  template <class COL, class COL_O> __global__ void
  halfTest(COL in, COL_O out, int strideI, int strideO)
  {
      const auto	row = blockIdx.y*blockDim.y + threadIdx.y;
      const auto	col = blockIdx.x*blockDim.x + threadIdx.x;
      
      in  += row*strideI + col;
      out += row*strideO + col;

      auto	val = *in;

      *out = val * val;
  }
}	// namespace device

template <class T> void
halfTest(const Array2<T>& in, Array2<T>& out)
{
    dim3	threads(in.ncol(), in.nrow());
    dim3	blocks(1, 1);
    device::halfTest<<<blocks, threads>>>(in.cbegin()->cbegin().get(),
					  out.begin()->begin().get(),
					  in.stride(), out.stride());
}
    
}	// namespace cu

template <class T> void
doJob()
{
    Array2<T>			a({{1.1f, 2.0f, 3.3f, 4.4f},
				   {5.5f, 6.6f, 7.7f, 8.8f}});
  //Array<T>		a({1, 2, 3, 4});
    std::cout << a;
    
    cu::Array2<__half>	in(a), out(in.nrow(), in.ncol());
    cu::halfTest(in, out);

    Array2<T>			b(out);
    std::cout << b;
}
}	// namespace TU

int
main()
{
    TU::doJob<float>();
    return 0;
}
