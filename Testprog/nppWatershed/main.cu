#include "hip/hip_runtime.h"
/*
 *  $Id: main.cc,v 1.1 2012-08-30 00:13:51 ueshiba Exp $
 */
#include "TU/Image++.h"
#include "TU/Profiler.h"
#include "TU/cuda/Array++.h"
#include "TU/cuda/algorithm.h"
#include "TU/cuda/chrono.h"
#include "TU/cuda/vec.h"
#include "TU/cuda/npp.h"

/************************************************************************
*  Global fucntions							*
************************************************************************/
int
main(int argc, char *argv[])
{
    using namespace	TU;

    using pixel_t	= uint8_t;
  //using pixel_t	= RGB;
    
    try
    {
	Image<pixel_t>	image;
	image.restore(std::cin);		// 原画像を読み込む

	cuda::Array2<pixel_t>	in_d(image), out_d(in_d.nrow(), in_d.ncol());
	cuda::nppiFilterGauss(in_d.cbegin(), in_d.cend(), out_d.begin(),
			      NPP_MASK_SIZE_15_X_15);

	Profiler<cuda::clock>	cuProfiler(1);
	constexpr size_t	NITER = 1000;

	for (size_t n = 0; n < NITER; ++n)
	{
	    cuProfiler.start(0);
	    cuda::nppiFilterGauss(in_d.cbegin(), in_d.cend(), out_d.begin(),
				  NPP_MASK_SIZE_15_X_15);
	    cuProfiler.nextFrame();
	}
	cuProfiler.print(std::cerr);

	image = out_d;
	image.save(std::cout);			// 結果画像をセーブ
    }
    catch (const std::exception& err)
    {
	std::cerr << err.what() << std::endl;
	return 1;
    }

    return 0;
}
