#include "hip/hip_runtime.h"
/*
 *  $Id$
 */
#include "TU/Profiler.h"
#include "TU/cuda/Array++.h"
#include "TU/cuda/functional.h"
#include "TU/cuda/algorithm.h"
#include "TU/cuda/chrono.h"

namespace TU
{
template <template <class> class OP, class S, class T> void
cudaJob(const Array2<S>& in, Array2<T>& out)
{
  // GPUによって計算する．
    cuda::Array2<S>	in_d(in);
    cuda::Array2<T>	out_d(in.nrow(), in.ncol());
    cuda::op3x3(in_d.cbegin(), in_d.cend(), out_d.begin(), OP<S>());
    hipDeviceSynchronize();

    Profiler<cuda::clock>	cuProfiler(1);
    constexpr size_t		NITER = 1000;
    for (size_t n = 0; n < NITER; ++n)		// フィルタリング
    {
	cuProfiler.start(0);
	cuda::op3x3(in_d.cbegin(), in_d.cend(), out_d.begin(), OP<S>());
	cuProfiler.nextFrame();
    }
    cuProfiler.print(std::cerr);
	
    out = out_d;
}

template void
cudaJob<cuda::maximal3x3>(const Array2<float>& in, Array2<float>& out)	;
}
