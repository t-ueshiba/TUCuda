#include "hip/hip_runtime.h"
/*
 *  $Id: main.cu,v 1.1 2012-08-30 00:13:51 ueshiba Exp $
 */
#include "TU/Image++.h"
#include "TU/Profiler.h"
#include "TU/cu/Array++.h"
#include "TU/cu/algorithm.h"
#include "TU/cu/functional.h"
#include "TU/cu/chrono.h"

//#define OP	cu::det3x3
//#define OP	cu::laplacian3x3
//#define OP	cu::sobelAbs3x3
#define OP	cu::maximal3x3
//#define OP	cu::minimal3x3

namespace TU
{
template <class T>
class maximal3x3
{
  public:
    using result_type = T;

    maximal3x3(T nonMaximal=0)	:_nonMaximal(nonMaximal)	{}

    template <class ITER> result_type
    operator ()(ITER p, ITER c, ITER n) const
    {
	return ((c[1] > p[0]) && (c[1] > p[1]) && (c[1] > p[2]) &&
		(c[1] > c[0])		       && (c[1] > c[2]) &&
		(c[1] > n[0]) && (c[1] > n[1]) && (c[1] > n[2]) ?
		c[1] : _nonMaximal);
    }

  private:
    const T	_nonMaximal;
};
}

/************************************************************************
*  Global fucntions							*
************************************************************************/
int
main(int argc, char *argv[])
{
    using namespace	std;
    using namespace	TU;

  //using in_t	= u_char;
  //using out_t	= u_char;
    using in_t	= float;
    using out_t	= float;
    
    try
    {
	Image<in_t>	in;
	in.restore(cin);				// 原画像を読み込む
	in.save(cout);					// 原画像をセーブ

      // GPUによって計算する．
	cu::Array2<in_t>	in_d(in);
	cu::Array2<out_t>	out_d(in.nrow(), in.ncol());
	cu::opNxM(in_d.cbegin(), in_d.cend(), out_d.begin(),
		  cu::maximal8<in_t>());
	hipDeviceSynchronize();

	Profiler<cu::clock>	cuProfiler(1);
	constexpr size_t	NITER = 1000;
	for (size_t n = 0; n < NITER; ++n)		// フィルタリング
	{
	    cuProfiler.start(0);
	    cu::opNxM(in_d.cbegin(), in_d.cend(), out_d.begin(),
		      cu::maximal8<in_t>());
	    cuProfiler.nextFrame();
	}
	cuProfiler.print(std::cerr);
	
	Image<out_t>	out(out_d);
	out.save(cout);					// 結果画像をセーブ

      // CPUによって計算する．
	Profiler<>	profiler(1);
	Image<out_t>	outGold;
	for (size_t n = 0; n < 10; ++n)
	{
	    outGold = in;
	    profiler.start(0);
	    op3x3(outGold.begin(), outGold.end(), TU::maximal3x3<in_t>());
	    profiler.nextFrame();
	}
	profiler.print(cerr);
	outGold.save(cout);

      // 結果を比較する．
	const int	V = 160;
	for (size_t u = 0; u < out.width(); ++u)
	    cerr << ' ' << (out[V][u] - outGold[V][u]);
	cerr <<  endl;
    }
    catch (exception& err)
    {
	cerr << err.what() << endl;
	return 1;
    }

    return 0;
}
