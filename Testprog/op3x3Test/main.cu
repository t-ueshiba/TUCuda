/*
 *  $Id: main.cc,v 1.1 2012-08-30 00:13:51 ueshiba Exp $
 */
#include <stdexcept>
#include "TU/Image++.h"
#include "TU/Profiler.h"
#include "TU/algorithm.h"
#include "TU/cuda/Array++.h"
#include "TU/cuda/functional.h"
#include "TU/cuda/algorithm.h"
#include <hip/hip_runtime.h>
#include <cutil.h>

//#define OP	cuda::det3x3
//#define OP	cuda::laplacian3x3
//#define OP	cuda::sobelAbs3x3
#define OP	cuda::maximal3x3
//#define OP	cuda::minimal3x3

/************************************************************************
*  Global fucntions							*
************************************************************************/
int
main(int argc, char *argv[])
{
    using namespace	std;
    using namespace	TU;

  //typedef u_char	in_t;
    typedef float	in_t;
  //typedef u_char	out_t;
    typedef float	out_t;
    
    try
    {
	Image<in_t>	in;
	in.restore(cin);				// 原画像を読み込む
	in.save(cout);					// 原画像をセーブ

      // GPUによって計算する．
	cuda::Array2<in_t>	in_d(in);
	cuda::Array2<out_t>	out_d(in.nrow(), in.ncol());

	u_int		timer = 0;
	CUT_SAFE_CALL(cutCreateTimer(&timer));		// タイマーを作成
	cuda::op3x3(in_d.cbegin(), in_d.cend(), out_d.begin(), OP<in_t>());
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUT_SAFE_CALL(cutStartTimer(timer));
	u_int	NITER = 1000;
	for (u_int n = 0; n < NITER; ++n)		// フィルタリング
	    cuda::op3x3(in_d.cbegin(), in_d.cend(), out_d.begin(), OP<in_t>());
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(cutStopTimer(timer));

	cerr << float(NITER * 1000) / cutGetTimerValue(timer) << "fps" << endl;
	CUT_SAFE_CALL(cutDeleteTimer(timer));		// タイマーを消去

	Image<out_t>	out;
	out_d.write(out);
	out.save(cout);					// 結果画像をセーブ
#if 1
      // CPUによって計算する．
	Profiler<>	profiler(1);
	Image<out_t>	outGold;
	for (u_int n = 0; n < 10; ++n)
	{
	    outGold = in;
	    profiler.start(0);
	  //op3x3(outGold.begin(), outGold.end(), OP<in_t, out_t>());
	    op3x3(outGold.begin(), outGold.end(), OP<in_t>());
	    profiler.stop().nextFrame();
	}
	profiler.print(cerr);
	outGold.save(cout);

      // 結果を比較する．
	const int	V = 160;
	for (u_int u = 0; u < out.width(); ++u)
	    cerr << ' ' << (out[V][u] - outGold[V][u]);
	cerr <<  endl;
#endif
    }
    catch (exception& err)
    {
	cerr << err.what() << endl;
	return 1;
    }

    return 0;
}
