#include "hip/hip_runtime.h"
/*
 *  $Id$
 */
#include "TU/Profiler.h"
#if 1
#  include "TU/cuda/BoxFilter.h"
#else
#  include "TU/cuda/NewBoxFilter.h"
#endif
#include "TU/cuda/functional.h"
#include "TU/cuda/chrono.h"

namespace TU
{
template <class T, class S> void
cudaJob(const Array2<T>& imageL, const Array2<T>& imageR, Array3<S>& scores,
	size_t winSize, size_t disparitySearchWidth)
{
  // スコアを計算する．
    cuda::BoxFilter2<S, 20>	cudaFilter(winSize, winSize);
    cuda::Array2<S>		imageL_d(imageL), imageR_d(imageR);
    cuda::Array3<S>		scores_d(32, imageL_d.nrow(), imageL_d.ncol(),
					 disparitySearchWidth);
    cudaFilter.convolve(imageL_d.cbegin(), imageL_d.cend(),
			imageR_d.cbegin(), scores_d.begin(),
			cuda::diff<T>(50), disparitySearchWidth);
    hipDeviceSynchronize();
#if 1
    Profiler<cuda::clock>	cudaProfiler(1);
    constexpr size_t		NITER = 100;
    for (size_t n = 0; n < NITER; ++n)
    {
	cudaProfiler.start(0);
	cudaFilter.convolve(imageL_d.cbegin(), imageL_d.cend(),
			    imageR_d.cbegin(), scores_d.begin(),
			    cuda::diff<T>(50), disparitySearchWidth);
	cudaProfiler.nextFrame();
    }
    cudaProfiler.print(std::cerr);
#endif
    
    scores = scores_d;;
}

template void
cudaJob(const Array2<u_char>& imageL, const Array2<u_char>& imageR,
	Array3<short>& out, size_t winSize, size_t disparitySearchWidth);
    
template void
cudaJob(const Array2<u_char>& imageL, const Array2<u_char>& imageR,
	Array3<float>& out, size_t winSize, size_t disparitySearchWidth);
    
}
