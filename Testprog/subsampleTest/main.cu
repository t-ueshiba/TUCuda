/*
 *  $Id: main.cc,v 1.1 2012-08-30 00:13:51 ueshiba Exp $
 */
#include <stdexcept>
#include "TU/Image++.h"
#include "TU/cuda/utility.h"
#include <hip/hip_runtime.h>
#include <cutil.h>

/************************************************************************
*  Global fucntions							*
************************************************************************/
int
main(int argc, char *argv[])
{
    using namespace	std;
    using namespace	TU;

    typedef float	pixel_t;
    
    try
    {
	Image<pixel_t>	image;
	image.restore(cin);				// 原画像を読み込む
	image.save(cout);
	
	CudaArray2<pixel_t>	in_d(image), out_d;
	
	u_int		timer = 0;
	CUT_SAFE_CALL(cutCreateTimer(&timer));		// タイマーを作成
	cuda::subsample(in_d, out_d);			// warp-up
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	u_int		NITER = 1000;
	CUT_SAFE_CALL(cutStartTimer(timer));
	for (u_int n = 0; n < NITER; ++n)
	    cuda::subsample(in_d, out_d);		// 実行
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(cutStopTimer(timer));

	cerr << float(NITER * 1000) / cutGetTimerValue(timer) << "fps" << endl;
	CUT_SAFE_CALL(cutDeleteTimer(timer));		// タイマーを消去

	out_d.write(image);
	image.save(cout);
    }
    catch (exception& err)
    {
	cerr << err.what() << endl;
	return 1;
    }

    return 0;
}
