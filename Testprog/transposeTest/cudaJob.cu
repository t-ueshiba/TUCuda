#include "hip/hip_runtime.h"
/*
 *  $Id$
 */
#include "TU/Profiler.h"
#include "TU/cuda/Array++.h"
#include "TU/cuda/algorithm.h"
#include "TU/cuda/chrono.h"

namespace TU
{
template <class S, class T> void
cudaJob(const Array2<S>& in, Array2<T>& out)
{
  // GPUによって計算する．
    cuda::Array2<S>	in_d(in);
    cuda::Array2<T>	out_d(in_d.ncol(), in_d.nrow());

    cuda::transpose(in_d.cbegin(), in_d.cend(), out_d.begin());
    hipDeviceSynchronize();

    Profiler<cuda::clock>	cuProfiler(1);
    constexpr size_t		NITER = 1000;
    for (size_t n = 0; n < NITER; ++n)
    {
	cuProfiler.start(0);
	cuda::transpose(in_d.cbegin(), in_d.cend(), out_d.begin());
	cuProfiler.nextFrame();
    }
    cuProfiler.print(std::cerr);
	
    out = out_d;
}

template void	cudaJob(const Array2<float>& in, Array2<float>& out)	;
}
