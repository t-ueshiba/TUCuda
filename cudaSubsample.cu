#include "hip/hip_runtime.h"
/*
 * $Id: cudaSubsample.cu,v 1.3 2011-04-19 04:00:25 ueshiba Exp $
 */
#include "TU/CudaUtility.h"

namespace TU
{
/************************************************************************
*  global constatnt variables						*
************************************************************************/
static const size_t	BlockDimX = 32;
static const size_t	BlockDimY = 16;
    
/************************************************************************
*  device functions							*
************************************************************************/
template <class T> static __global__ void
subsample_kernel(const T* in, T* out, uint stride_i, uint stride_o)
{
    const int	tx = threadIdx.x,
		ty = threadIdx.y;
    const int	bw = blockDim.x;
    const int	x0 = blockIdx.x*bw,
		y  = blockIdx.y*blockDim.y + ty;
    const int	xy = 2*(y*stride_i + x0)   + tx;
    
  // ��������2x2�֥�å���1�Ԥ����˶�ͭ����˥��ԡ�
    __shared__ T	in_s[BlockDimY][2*BlockDimX+1];
    in_s[ty][tx	    ] = in[xy     ];
    in_s[ty][tx + bw] = in[xy + bw];
    __syncthreads();

    out[y*stride_o + x0 + tx] = in_s[ty][2*tx];
}

/************************************************************************
*  global functions							*
************************************************************************/
//! CUDA�ˤ�ä�2����������ʿ����ľ�������줾��1/2�˴ְ�����
/*!
  \param in	����2��������
  \param out	����2��������
*/
template <class T> void
cudaSubsample(const CudaArray2<T>& in, CudaArray2<T>& out)
{
    out.resize(in.nrow()/2, in.ncol()/2);

    dim3	threads(BlockDimX, BlockDimY);
    dim3	blocks(out.ncol() / threads.x, out.nrow() / threads.y);

  // ����
    subsample_kernel<<<blocks, threads>>>(in.data(), out.data(),
					  in.stride(), out.stride());

  // ��ü
    uint	bottom = blocks.y * threads.y;
    threads.y = out.nrow() % threads.y;
    blocks.y  = 1;
    subsample_kernel<<<blocks, threads>>>(in.data()  + bottom * in.stride(),
					  out.data() + bottom * out.stride(),
					  in.stride(), out.stride());

  // ��ü
    uint	right = blocks.x * threads.x;
    threads.x = out.ncol() % threads.x;
    blocks.x  = 1;
    threads.y = BlockDimY;
    blocks.y  = out.nrow() / threads.y;
    subsample_kernel<<<blocks, threads>>>(in.data()  + right,
					  out.data() + right,
					  in.stride(), out.stride());

  // ����
    threads.y = out.nrow() % threads.y;
    blocks.y  = 1;
    subsample_kernel<<<blocks, threads>>>(in.data()  + bottom * in.stride()
						     + right,
					  out.data() + bottom * out.stride()
						     + right,
					  in.stride(), out.stride());
}

template void	cudaSubsample(const CudaArray2<u_char>& in,
				    CudaArray2<u_char>& out)	;
template void	cudaSubsample(const CudaArray2<short>& in,
				    CudaArray2<short>& out)	;
template void	cudaSubsample(const CudaArray2<int>& in,
				    CudaArray2<int>& out)	;
template void	cudaSubsample(const CudaArray2<float>& in,
				    CudaArray2<float>& out)	;
}
